#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <string.h>

#include "util.h"
#include "ppm.h"

#define IMAGE_DIM 2048
#define SAMPLE_SIZE 10
#define SAMPLE_DIM (SAMPLE_SIZE*2+1)
#define NUMBER_OF_SAMPLES (SAMPLE_DIM*SAMPLE_DIM)


/************************************/
/* Kernel for blur using GPU memory */
/************************************/
__global__ void image_blur(uchar4 *image, uchar4 *image_output) {
    // TODO calculate array index based on special CUDA variables
    uint i = blockIdx.x * blockDim.x + threadIdx.x;
    uint j = blockIdx.y * blockDim.y + threadIdx.y;

    int blur_radius = 10;
    int blur_area = (2*blur_radius+1)*(2*blur_radius+1);

    int r = 0, g = 0, b = 0;

    // TODO loop through adjacent pixel coordinates
    for (int x_offset = -blur_radius; x_offset <= blur_radius; x_offset++) {
        for (int y_offset = -blur_radius; y_offset <= blur_radius; y_offset++) {
            // TODO calculate index of adjacent pixel
            int x = i + x_offset;
            int y = j + y_offset;
            if (x < 0 || x >= IMAGE_DIM) {
                 /*wrap the blur area*/
                x = (x % IMAGE_DIM + IMAGE_DIM) % IMAGE_DIM;
            }
            if (y < 0 || y >= IMAGE_DIM) {
                 /*wrap the blur area*/
                y = (y % IMAGE_DIM + IMAGE_DIM) % IMAGE_DIM;
            }
            // TODO read pixels from row-major array
            uchar4 pixel = image[x + y * IMAGE_DIM];
            r += pixel.x;
            g += pixel.y;
            b += pixel.z;
        }
    }
    // TODO calculate average of surrounding pixels
    uchar4 avg_pixel;
    avg_pixel.x = r / blur_area;
    avg_pixel.y = g / blur_area;
    avg_pixel.z = b / blur_area;

    // TODO store averaged pixel in image_output
    image_output[i + j * IMAGE_DIM] = avg_pixel;
}


/************************************/
/* Kernel for blur using 1D texture */
/************************************/
__global__ void image_blur_texture1D(hipTextureObject_t sample1D, uchar4 *image_output) {
    // TODO calculate array index based on special CUDA variables
    uint i = blockIdx.x * blockDim.x + threadIdx.x;
    uint j = blockIdx.y * blockDim.y + threadIdx.y;

    int blur_radius = 10;
    int blur_area = (2*blur_radius+1)*(2*blur_radius+1);

    int r = 0, g = 0, b = 0;

    // TODO loop through adjacent pixel coordinates
    for (int x_offset = -blur_radius; x_offset <= blur_radius; x_offset++) {
        for (int y_offset = -blur_radius; y_offset <= blur_radius; y_offset++) {
            // TODO calculate index of adjacent pixel
            int x = i + x_offset;
            int y = j + y_offset;
            if (x < 0 || x >= IMAGE_DIM) {
                /* wrap the blur area */
                x = (x % IMAGE_DIM + IMAGE_DIM) % IMAGE_DIM;
            }
            if (y < 0 || y >= IMAGE_DIM) {
                /* wrap the blur area */
                y = (y % IMAGE_DIM + IMAGE_DIM) % IMAGE_DIM;
            }
            // TODO read pixels from row-major array
            uchar4 pixel = tex1Dfetch<uchar4>(sample1D, (float)(x + y * IMAGE_DIM));
            r += pixel.x;
            g += pixel.y;
            b += pixel.z;
        }
    }

    // TODO calculate average of surrounding pixels
    uchar4 avg_pixel;
    avg_pixel.x = r / blur_area;
    avg_pixel.y = g / blur_area;
    avg_pixel.z = b / blur_area;

    // TODO store averaged pixel in image_output
    image_output[i + j * IMAGE_DIM] = avg_pixel;
}


/************************************/
/* Kernel for blur using 2D texture */
/************************************/
__global__ void image_blur_texture2D(hipTextureObject_t sample2D, uchar4 *image_output) {
    uint i = blockIdx.x * blockDim.x + threadIdx.x;
    uint j = blockIdx.y * blockDim.y + threadIdx.y;

    int blur_radius = 10;
    int blur_area = (2*blur_radius+1)*(2*blur_radius+1);

    int r = 0, g = 0, b = 0;

    // TODO loop through adjacent pixel coordinates
    for (int x_offset = -blur_radius; x_offset <= blur_radius; x_offset++) {
        for (int y_offset = -blur_radius; y_offset <= blur_radius; y_offset++) {
            // TODO calculate index of adjacent pixel
            int x = i + x_offset;
            int y = j + y_offset;
            if (x < 0 || x >= IMAGE_DIM) {
                /* wrap the blur area */
                x = (x % IMAGE_DIM + IMAGE_DIM) % IMAGE_DIM;
            }
            if (y < 0 || y >= IMAGE_DIM) {
                /* wrap the blur area */
                y = (y % IMAGE_DIM + IMAGE_DIM) % IMAGE_DIM;
            }
            // TODO read pixels from row-major array
            uchar4 pixel = tex2D<uchar4>(sample2D, (float)x, (float)y);
            r += pixel.x;
            g += pixel.y;
            b += pixel.z;
        }
    }

    // TODO calculate average of surrounding pixels
    uchar4 avg_pixel;
    avg_pixel.x = r / blur_area;
    avg_pixel.y = g / blur_area;
    avg_pixel.z = b / blur_area;

    // TODO store averaged pixel in image_output
    image_output[i + j * IMAGE_DIM] = avg_pixel;
}


int main(int argc, char **argv) {
    unsigned int image_size;
    uchar4 *d_image, *d_image_output;
    uchar4 *h_image;
    hipEvent_t start, stop;

    image_size = IMAGE_DIM * IMAGE_DIM * sizeof(uchar4);

    if (argc != 3) {
        printf("Syntax: %s mode outputfilename.ppm\n\twhere mode is 0, 1, or 2\n", argv[0]);
        exit(1);
    }
    int mode = atoi(argv[1]);
    const char *filename = argv[2];

    // create timers
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // allocate memory on the GPU for the output image
    CHECK_ERROR(hipMalloc((void**)&d_image, image_size));
    CHECK_ERROR(hipMalloc((void**)&d_image_output, image_size));

    // allocate and load host image
    h_image = (uchar4*)malloc(image_size);
    if (h_image == NULL) {
        printf("Malloc failed");
        exit(1);
    }
    input_image_file("input.ppm", h_image, IMAGE_DIM);

    // copy image to device memory
    CHECK_ERROR(hipMemcpy(d_image, h_image, image_size, hipMemcpyHostToDevice));

    //cuda layout and execution
    dim3    blocksPerGrid(IMAGE_DIM / 16, IMAGE_DIM / 16);
    dim3    threadsPerBlock(16, 16);

    switch (mode) {


        /*************************/
        /* Blur using GPU memory */
        /*************************/
        case 0:
        {
            // normal version
            hipEventRecord(start, 0);
            image_blur<<<blocksPerGrid, threadsPerBlock>>>(d_image, d_image_output);
            check_launch("kernel normal");
            hipEventRecord(stop, 0);
            hipEventSynchronize(stop);
            float ms;
            hipEventElapsedTime(&ms, start, stop);
            printf("Blur using device memory, time: %f\n", ms);
        }
        break;


        /*************************/
        /* Blur using 1D texture */
        /*************************/
        case 1:
        {
            hipTextureObject_t sample1d=0;

            /*
                We use this hipResourceDesc to describe the
                structure of our data so CUDA can do indexing for us.
                The relevant details: linear array, of unsigned values,
                such that every element consists of four 8-bit values.
            */
            hipResourceDesc resDesc;
            memset(&resDesc, 0, sizeof(resDesc));
            resDesc.resType = hipResourceTypeLinear;
            resDesc.res.linear.devPtr = d_image;

            resDesc.res.linear.desc.f = hipChannelFormatKindUnsigned;
            resDesc.res.linear.desc.x = resDesc.res.linear.desc.y = 
                resDesc.res.linear.desc.z = resDesc.res.linear.desc.w = 8; // bits per channel
            resDesc.res.linear.sizeInBytes = image_size;

            hipTextureDesc texDesc;
            memset(&texDesc, 0, sizeof(texDesc));
            texDesc.readMode = hipReadModeElementType;

            CHECK_ERROR(hipCreateTextureObject(&sample1d, &resDesc, &texDesc, NULL));

            hipEventRecord(start, 0);
            image_blur_texture1D<<<blocksPerGrid, threadsPerBlock>>>(sample1d, d_image_output);
            check_launch("kernel tex1D");
            hipEventRecord(stop, 0);
            hipEventSynchronize(stop);
            float ms;
            hipEventElapsedTime(&ms, start, stop);
            hipDestroyTextureObject(sample1d);
            printf("Blur using 1D texture, time: %f\n", ms);
        }
        break;

        /*************************/
        /* Blur using 2D texture */
        /*************************/
        case 2:
        {
            hipTextureObject_t sample2d=0;

            /*
                We use this hipResourceDesc to describe the
                structure of our data so CUDA can do indexing for us.
                The relevant details: pitched-2D array with no padding
                between rows, of unsigned values, where each array element
                consists of four 8-bit values, and there are a total of
                IMAGE_DIM * IMAGE_DIM elements.
            */
            hipResourceDesc resDesc;
            memset(&resDesc, 0, sizeof(resDesc));
            resDesc.resType = hipResourceTypePitch2D;

            resDesc.res.pitch2D.devPtr = d_image;
            resDesc.res.pitch2D.desc.f = hipChannelFormatKindUnsigned;
            resDesc.res.pitch2D.desc.x = resDesc.res.linear.desc.y = 
                resDesc.res.linear.desc.z = resDesc.res.linear.desc.w = 8; // bits per channel
            resDesc.res.pitch2D.width = IMAGE_DIM;
            resDesc.res.pitch2D.height = IMAGE_DIM;
            resDesc.res.pitch2D.pitchInBytes = IMAGE_DIM * sizeof(uchar4);

            hipTextureDesc texDesc;
            memset(&texDesc, 0, sizeof(texDesc));
            texDesc.readMode = hipReadModeElementType;

            CHECK_ERROR(hipCreateTextureObject(&sample2d, &resDesc, &texDesc, NULL));

            hipEventRecord(start, 0);
            image_blur_texture2D<<<blocksPerGrid, threadsPerBlock>>>(sample2d, d_image_output);
            check_launch("kernel tex2D");
            hipEventRecord(stop, 0);
            hipEventSynchronize(stop);
            float ms;
            hipEventElapsedTime(&ms, start, stop);
            hipDestroyTextureObject(sample2d);
            printf("Blur using 2D texture, time: %f\n", ms);
        }
        break;

        default:
            printf("Unknown mode %d\n", mode);
            exit(1);
            break;
    }


    // copy the image back from the GPU for output to file
    CHECK_ERROR(hipMemcpy(h_image, d_image_output, image_size, hipMemcpyDeviceToHost));

    // output image
    output_image_file(filename, h_image, IMAGE_DIM);

    //cleanup
    hipEventDestroy(start);
    hipEventDestroy(stop);
    hipFree(d_image);
    hipFree(d_image_output);
    free(h_image);

    return 0;
}
